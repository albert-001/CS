
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

//Hillis Steele scan in one block;
__global__ void prefixOnDevice(int *a, int *b, int n){
	int id = threadIdx.x;
	int *s;
	for(int j=1; j<n; j<<=1){
		if(id >=j)
			b[id] = a[id-j] + a[id];
		else
			b[id] = a[id];
		s = a;
		a = b;
		b = s;
		__syncthreads();
	}	
}

void prefixOnHost(int *a, int n){
	int sum =0;
	for(int i=0; i<n; i++){
		sum += a[i];
		a[i] = sum;
	}
}

//one value per thread, with power of two number of threads
int main(int argc, char **argv){
	int blockSize = 128;
	int nBlocks = 1;
	int n = blockSize;
	int size = n*sizeof(int);
	int *a = (int*) malloc(size);
	int *b = (int*) malloc(size);
	int *a_d, *b_d;
	hipMalloc((void**) &a_d, size);
	hipMalloc((void**) &b_d, size);
	for(int i=0; i<n; i++)
		a[i] = i+1;
	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

	prefixOnHost(a, n);
	
	prefixOnDevice <<<nBlocks, blockSize>>> (a_d, b_d, n);

	if((int)log2((double)n)%2 == 0)
		hipMemcpy(b, a_d, size, hipMemcpyDeviceToHost);
	else
		hipMemcpy(b, b_d, size, hipMemcpyDeviceToHost);

	for(int i=0; i<n; i++)
		//printf("%d %d\n", a[i], b[i]);
		assert(a[i] == b[i]);
	return 0;
}
