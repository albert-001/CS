#include "hip/hip_runtime.h"
/* Copyright 2017 Eric Aubanel
 * This file contains code implementing Algorithm 4.14 from
 * Elements of Parallel Computing, by Eric Aubanel, 2016, CRC Press.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * -------------------------------------------------------------------
 * Implementation of reduction of n floats using CUDA
 */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int isPowerOf2(int n);

//reduce n floats in array a to a partial sum for each block,
//stored in array c. Block size must be power of 2
__global__ void reductionGPU(float *a, float *c, int n){
	//size of b indicated by kernel call in main (blockSize)
  extern __shared__ float b[];
	int gsize = blockDim.x; //block size
	int nt = gsize * gridDim.x; //total number of threads
	int gid = blockIdx.x; //block id
	int tid = threadIdx.x; //local thread id
	int id = gid*gsize + tid; //global thread id

	//if n<nt, then some threads will have nothing to do
	int istart;
	int iend;
	if(n<nt){
	    istart = id;
	    iend = id;
	}
	else{
		//evaluate as float to avoid overflow
		istart = (float)id*n/nt;
		iend = (float)(id+1)*n/nt - 1;	
	}
	if(n<nt && id>=n){
		b[tid] = 0;
	}else{
		float psum = 0.0;
		for(int i=istart; i<=iend; i++)
			psum += a[i];
		b[tid] = psum;
	}

    __syncthreads();
	for(int j=gsize>>1; j>=1; j >>= 1){
		if(tid<j)
			b[tid] += b[tid+j];
		__syncthreads();
	}
	c[gid] = b[0];
}

int main(int argc, char **argv){
	float *a_h; //array to be reduced on host (CPU)
	float *c_h; //array of partial sums on host
	float *a_d; //array to be reduced on device (GPU)
	float *c_d; //array of partial sums on device
	hipError_t error1, error2;
	struct timespec tstart, tend;
	float time;

	if(argc < 4){
		fprintf(stderr,"usage: %s n blockSize numBlocks\n", argv[0]);
		return 1;
	}
	int n = strtol(argv[1], NULL, 10);
	int blockSize = strtol(argv[2], NULL, 10); //size of thread block on device
	int numBlocks = strtol(argv[3], NULL, 10); //number of blocks on device
	if(!isPowerOf2(blockSize)){
		fprintf(stderr,"blockSize must be power of 2\n");
		return 1;
	}

	//memory allocation on host and device
	a_h = (float *)malloc(n*sizeof(float));
	c_h = (float *)malloc(numBlocks*sizeof(float));
	if(a_h == NULL || c_h == NULL){
		fprintf(stderr,"couldn't allocate memory on host\n");
		return 1;
	}
	error1 = hipMalloc((void **)&a_d, n*sizeof(float));
	error2 = hipMalloc((void **)&c_d, numBlocks*sizeof(float));
	if(error1 != hipSuccess || error2 != hipSuccess){
		fprintf(stderr,"couldn't allocate memory on device\n");
		return 1;
	}

	for(int i=0; i<n; i++)
		a_h[i] = rand()%100;

	//sequential reduction for verification and timing
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	float sum = 0.0;
	for(int i=0; i<n; i++)
		sum += a_h[i];
	clock_gettime(CLOCK_MONOTONIC, &tend);
	time = (tend.tv_sec-tstart.tv_sec) + (tend.tv_nsec-tstart.tv_nsec)*1.0e-9;
	printf("CPU reduction time in s: %f\n", time);

	//timing won't include transfer of array to device
	hipMemcpy(a_d, a_h, n*sizeof(float), hipMemcpyHostToDevice);
	clock_gettime(CLOCK_MONOTONIC, &tstart);
	reductionGPU <<<numBlocks, blockSize, blockSize*sizeof(float)>>> (a_d, c_d, n);
	error1 = hipDeviceSynchronize();// wait until GPU kernel finished
	if(error1 != hipSuccess){
		fprintf(stderr,"error executing kernel: %s\n", hipGetErrorString(error1));
		return 1;
	}

	//Do not copy back to host and do CPU reduction. Do it on the GPU recersively.
	while(1){
		hipMemset(a_d, 0, n);
		n = numBlocks;
		numBlocks = n/blockSize + ((n%blockSize==0)?0:1);
		hipMemcpy(a_d, c_d, n*sizeof(float), hipMemcpyDeviceToDevice);
		hipMemset(c_d, 0, n*sizeof(float));
		reductionGPU <<<numBlocks, blockSize, blockSize*sizeof(float)>>> (a_d, c_d, n);
		error2 = hipDeviceSynchronize();// wait until GPU kernel finished
		if(error2 != hipSuccess){
			fprintf(stderr,"error executing kernel: %s\n", hipGetErrorString(error2));
			return 1;
		}
		hipMemcpy(c_h, c_d, numBlocks*sizeof(float), hipMemcpyDeviceToHost);	
		if(numBlocks==1)break;
	}
	hipMemcpy(c_h, c_d, numBlocks*sizeof(float), hipMemcpyDeviceToHost);
	float dsum = c_h[0];
	printf("dsum: %f\n", dsum);

	clock_gettime(CLOCK_MONOTONIC, &tend);
	time = (tend.tv_sec-tstart.tv_sec) + (tend.tv_nsec-tstart.tv_nsec)*1.0e-9;
	printf("GPUS time in s: %f\n", time);

	//not necessarily the same because of differences in roundoff error
	printf("relative difference between sequential and parallel sums: %g\n",
			fabs(dsum-sum)/sum);
	return 0;
}

int isPowerOf2(int n){
	while(n){
		if(n & 1)
			break;
		n >>= 1;
	}
	return (1 == n? 1:0);
}